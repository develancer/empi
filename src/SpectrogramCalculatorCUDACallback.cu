/**********************************************************
 * Piotr T. Różański (c) 2015-2021                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include "CUDA.h"

static __device__  real myInputCallback(void *dataIn, size_t offset, void *callerInfo, void *) {
    auto *info = reinterpret_cast<CudaCallbackInfo *>(callerInfo);
    size_t index_in_batch = offset & info->window_length_mask;
    real ret = (index_in_batch < info->envelope_length)
               ? ((real *) dataIn)[(offset >> info->window_length_bits) * info->input_shift + index_in_batch] * info->envelope[index_in_batch]
               : 0;
    return ret;
}

static __device__  void
myOutputCallback(void *dataOut, size_t offset, cucomplex element, void *callerInfo, void *) {
    auto *info = reinterpret_cast<CudaCallbackInfo *>(callerInfo);
    size_t index_in_batch = offset % info->spectrum_length;
    if (index_in_batch < info->output_bins) {
        ((cucomplex *) dataOut)[offset / info->spectrum_length * info->output_bins + index_in_batch] = element;
    }
}

static __device__ hipfftCallbackLoadD myInputPtr = myInputCallback;

static __device__ hipfftCallbackStoreZ myOutputPtr = myOutputCallback;

void CudaCallback::initialize() {
    cuda_check(hipMemcpyFromSymbol(&hostCopyOfInputCallback, HIP_SYMBOL(myInputPtr), sizeof(void*)));
    cuda_check(hipMemcpyFromSymbol(&hostCopyOfOutputCallback, HIP_SYMBOL(myOutputPtr), sizeof(void*)));
}

void CudaCallback::associate(hipfftHandle plan, CudaCallbackInfo *dev_info) {
    cufft_check(hipfftXtSetCallback(plan, (void **) &hostCopyOfInputCallback, HIPFFT_CB_LD_REAL_DOUBLE, (void **) &dev_info));
    cufft_check(hipfftXtSetCallback(plan, (void **) &hostCopyOfOutputCallback, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void **) &dev_info));
}
