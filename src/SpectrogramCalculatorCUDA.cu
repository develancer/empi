#include "hip/hip_runtime.h"
/**********************************************************
 * Piotr T. Różański (c) 2015-2023                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include <algorithm>
#include <list>
#include <map>
#include <memory>
#include <set>
#include <utility>
#include "Array.h"
#include "Corrector.h"
#include "CUDA.h"
#include "IndexRange.h"
#include "PinnedArray.h"
#include "SpectrogramCalculatorCUDA.h"

char CudaException::buffer[256];

void cuda_check(hipError_t error) {
    if (error) {
        if (error == hipErrorOutOfMemory) {
            throw CudaMemoryException();
        }
        throw CudaException(error);
    }
}

void cufft_check(hipfftResult_t result) {
    if (result) {
        if (result == HIPFFT_ALLOC_FAILED) {
            throw CudaMemoryException();
        }
        throw CudaException(result);
    }
}

//////////////////////////////////////////////////////////////////////////////

class CudaCorrector {
    cucomplex ft;
    cucomplex re_factor;
    cucomplex im_factor;

public:
    CudaCorrector() = delete;

    __device__ double compute(cucomplex value) const {
        const cucomplex corrected = make_hipDoubleComplex(
                value.x * re_factor.x + value.y * im_factor.x,
                value.x * re_factor.y + value.y * im_factor.y
        );
        const cucomplex corrected2 = hipCmul(corrected, corrected);
        const double norm_corrected = corrected.x * corrected.x + corrected.y * corrected.y;
        return 0.5 * (norm_corrected + corrected2.x * ft.x + corrected2.y * ft.y);
    }
} __attribute__ ((aligned (16)));

//////////////////////////////////////////////////////////////////////////////

static __device__ __inline__ void oneReduce(volatile ExtractedMaximum &dest, const volatile ExtractedMaximum &other) {
    if (other.energy > dest.energy) {
        dest.energy = other.energy;
        dest.bin_index = other.bin_index;
    }
}

template<unsigned THREADS_IN_BLOCK>
static __device__ __inline__ void warpReduce(volatile ExtractedMaximum *sdata, unsigned tid) {
    if (THREADS_IN_BLOCK >= 64) oneReduce(sdata[tid], sdata[tid + 32]);
    if (THREADS_IN_BLOCK >= 32) oneReduce(sdata[tid], sdata[tid + 16]);
    if (THREADS_IN_BLOCK >= 16) oneReduce(sdata[tid], sdata[tid + 8]);
    if (THREADS_IN_BLOCK >= 8) oneReduce(sdata[tid], sdata[tid + 4]);
    if (THREADS_IN_BLOCK >= 4) oneReduce(sdata[tid], sdata[tid + 2]);
    if (THREADS_IN_BLOCK >= 2) oneReduce(sdata[tid], sdata[tid + 1]);
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorSingleChannelCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;
    assert(channel_count == 1);

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        double energy = correctors[i].compute(spectra[0][offset + i]);
        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

/**
 * @param z must be of amplitude 1
 * @return square root of z
 */
static __device__ cucomplex cuCsqrtUnit(cucomplex z) {
    const double abs_z_plus_r = hypot(hipCreal(z) + 1.0, hipCimag(z));
    if (abs_z_plus_r > 1.0e-12) {
        const double scale = 1.0 / abs_z_plus_r;
        return make_hipDoubleComplex(scale * (hipCreal(z) + 1.0), scale * hipCimag(z));
    } else {
        // z is equal to -1
        return make_hipDoubleComplex(0.0, 1.0);
    }
}

static inline __device__ double cuCnorm(cucomplex z) {
    const double real = hipCreal(z), imag = hipCimag(z);
    return real * real + imag * imag;
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorConstantPhaseCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        cucomplex direction = make_hipDoubleComplex(0, 0);
        for (unsigned c = 0; c < channel_count; ++c) {
            direction = hipCfma(spectra[c][offset + i], spectra[c][offset + i], direction);
        }
        double abs_direction = hipCabs(direction);
        cucomplex best_direction = (abs_direction > 0)
                                   ? cuCsqrtUnit(make_hipDoubleComplex(hipCreal(direction) / abs_direction, hipCimag(direction) / abs_direction))
                                   : make_hipDoubleComplex(1.0, 0.0);
        double energy_correction = correctors[i].compute(best_direction);

        double energy = 0;
        for (unsigned c = 0; c < channel_count; ++c) {
            const cucomplex v = spectra[c][offset + i];
            double norm_channel = cuCnorm(v);
            if (norm_channel > 0) {
                double cos_factor = (hipCreal(v) * hipCreal(best_direction) + hipCimag(v) * hipCimag(best_direction))
                                    / sqrt(norm_channel);
                energy += norm_channel * cos_factor * cos_factor;
            }
        }
        energy *= energy_correction;

        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorVariablePhaseCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        real energy = 0;
        for (unsigned c = 0; c < channel_count; ++c) {
            const cucomplex v = spectra[c][offset + i];
            energy += correctors[i].compute(v);
        }
        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

void callExtractorKernel(const SpectrogramRequest &request, hipStream_t stream, cucomplex **spectra,
                         ExtractedMaximum *maxima, Corrector *correctors) {
    const unsigned threads_in_block = 64;
    if (request.extractor == extractorVariablePhase) {
        extractorVariablePhaseCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
                (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else if (request.extractor == extractorConstantPhase) {
        extractorConstantPhaseCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
                (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else if (request.extractor == extractorSingleChannel) {
        extractorSingleChannelCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
                (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else {
        fprintf(stderr, "ERROR: unsupported extractor\n");
        exit(1);
    }
}

void *cuda_dev_malloc(size_t length) {
    void *result;
    cuda_check(hipMalloc(&result, length));
    return result;
}

void cuda_dev_free(void *pointer) {
    cuda_check(hipFree(pointer));
}

template<typename T>
T *cuda_dev_alloc(size_t length) {
    return reinterpret_cast<T *>(cuda_dev_malloc(sizeof(T) * length));
}

template<typename T>
Array1D<T> cuda_dev_array_1d(size_t n) {
    return Array1D<T>(n, cuda_dev_alloc<T>, cuda_dev_free);
}

template<typename T>
Array2D<T> cuda_dev_array_2d(size_t n, size_t m) {
    return Array2D<T>(n, m, cuda_dev_alloc<T>, cuda_dev_free);
}

std::shared_ptr<ihipStream_t> cuda_create_stream() {
    hipStream_t stream;
    cuda_check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    return {stream, hipStreamDestroy};
}

//////////////////////////////////////////////////////////////////////////////

class CudaTask {
    const int device;
    std::map<std::pair<int, int>, hipfftHandle> plans;
    CudaCallback callback;

    std::shared_ptr<CudaCallbackInfo> info;
    std::shared_ptr<CudaCallbackInfo> dev_info;
    Array1D<real> dev_envelope;
    Array1D<Corrector> dev_correctors;

    std::shared_ptr<ihipStream_t> stream;

    Array1D<char> dev_workspace;
    Array1D<real> dev_input;
    Array2D<cucomplex> outputs;
    Array1D<cucomplex *> dev_outputs;
    Array1D<ExtractedMaximum> dev_maxima;

    void process(const SpectrogramRequest &request, hipfftHandle handle) {
        index_t total_input_length = request.envelope_length + (request.how_many - 1) * request.input_shift;
        IndexRange overlap = IndexRange(-request.input_offset, request.channel_length - request.input_offset).overlap(total_input_length);
        if (!overlap) {
            // all samples consist of zero-padding
            std::fill(request.maxima, request.maxima + request.how_many, ExtractedMaximum{0, 0});
            return;
        }

        for (int c = 0; c < request.channel_count; ++c) {
            if (overlap.first_index) {
                cuda_check(hipMemsetAsync(dev_input.get(), 0, overlap.first_index * sizeof(real), stream.get()));
            }
            cuda_check(hipMemcpyAsync(dev_input.get() + overlap.first_index,
                                       request.data[c] + request.input_offset + overlap.first_index,
                                       (overlap.end_index - overlap.first_index) * sizeof(real), hipMemcpyHostToDevice,
                                       stream.get()));
            if (overlap.end_index < total_input_length) {
                cuda_check(hipMemsetAsync(dev_input.get() + overlap.end_index, 0,
                                           (total_input_length - overlap.end_index) * sizeof(real), stream.get()));
            }
            cufft_check(hipfftExecD2Z(handle, dev_input.get(), outputs[c]));
        }
        callExtractorKernel(request, stream.get(), dev_outputs.get(), dev_maxima.get(), dev_correctors.get());
        cuda_check(hipMemcpyAsync(request.maxima, dev_maxima.get(), request.how_many * sizeof(ExtractedMaximum),
                                   hipMemcpyDeviceToHost, stream.get()));
    }

public:
    CudaTask(int channel_count, const std::list<ProtoRequest> &proto_requests, int device) : device(device) {
        int max_envelope_length = 0;
        int max_output_bins = 0;
        int max_window_length = 0;
        for (const auto &proto_request : proto_requests) {
            max_envelope_length = std::max(max_envelope_length, proto_request.envelope_length);
            max_output_bins = std::max(max_output_bins, proto_request.output_bins);
            max_window_length = std::max(max_window_length, proto_request.window_length);
        }

        cuda_check(hipSetDevice(device));
        callback.initialize();
        stream = cuda_create_stream();

        info.reset(cuda_host_alloc<CudaCallbackInfo>(1), cuda_host_free);
        dev_info.reset(cuda_dev_alloc<CudaCallbackInfo>(1), cuda_dev_free);
        dev_envelope = cuda_dev_array_1d<real>(max_envelope_length);
        dev_correctors = cuda_dev_array_1d<Corrector>(max_output_bins);
        dev_outputs = cuda_dev_array_1d<cucomplex *>(channel_count);

        size_t free_memory, total_memory;
        cuda_check(hipMemGetInfo(&free_memory, &total_memory));

        size_t max_dev_input_length = 0;
        size_t max_dev_output_length = 0;
        size_t max_work_size = 0;
        int max_how_many = 0;
        std::map<int, std::set<int>> how_many_for_various_windows;
        for (const auto &proto_request : proto_requests) {
            int window_length = proto_request.window_length;
            int spectrum_length = window_length / 2 + 1;

            for (int how_many = 1; how_many <= proto_request.how_many; how_many *= 2) {
                size_t dev_input_length = static_cast<size_t>(proto_request.envelope_length)
                                          + static_cast<size_t>(how_many - 1) * static_cast<size_t>(proto_request.input_shift);
                size_t dev_output_length = static_cast<size_t>(how_many) * static_cast<size_t>(proto_request.output_bins);
                size_t work_size;
                cufft_check(hipfftEstimateMany(
                        1, &window_length,
                        &window_length, 1, window_length,
                        &spectrum_length, 1, spectrum_length,
                        HIPFFT_D2Z, how_many, &work_size
                ));

                size_t memory_needed = work_size
                                       + sizeof(double) * dev_input_length
                                       + sizeof(cucomplex) * static_cast<size_t>(channel_count) * dev_output_length
                                       + sizeof(ExtractedMaximum) * static_cast<size_t>(how_many);
                if (memory_needed > free_memory) {
                    if (how_many == 1) {
                        throw std::runtime_error("not enough CUDA memory");
                    }
                    break;
                }

                max_dev_input_length = std::max(max_dev_input_length, dev_input_length);
                max_dev_output_length = std::max(max_dev_output_length, dev_output_length);
                max_work_size = std::max(max_work_size, work_size);
                max_how_many = std::max(max_how_many, how_many);

                how_many_for_various_windows[window_length].insert(how_many);
            }
        }

        dev_input = cuda_dev_array_1d<double>(max_dev_input_length);

        outputs = cuda_dev_array_2d<cucomplex>(channel_count, max_dev_output_length);
        cuda_check(hipMemcpy(dev_outputs.get(), outputs.get(), channel_count * sizeof(cucomplex *), hipMemcpyHostToDevice));

        dev_maxima = cuda_dev_array_1d<ExtractedMaximum>(max_how_many);

        max_work_size = 0; // this will now be calculated more precisely
        for (const auto &pair : how_many_for_various_windows) {
            int window_length = pair.first;
            int spectrum_length = window_length / 2 + 1;
            for (int how_many : pair.second) {
                hipfftHandle handle;
                size_t work_size = 0;
                cufft_check(hipfftCreate(&handle));
                cufft_check(hipfftSetAutoAllocation(handle, 0));
                cufft_check(hipfftMakePlanMany(
                        handle, 1, &window_length,
                        &window_length, 1, window_length,
                        &spectrum_length, 1, spectrum_length,
                        HIPFFT_D2Z, how_many, &work_size
                ));
                max_work_size = std::max(max_work_size, work_size);
                callback.associate(handle, dev_info.get());
                cufft_check(hipfftSetStream(handle, stream.get()));
                plans.emplace(std::make_pair(window_length, how_many), handle);
            }
        }

        dev_workspace = cuda_dev_array_1d<char>(max_work_size);
        for (auto &pair : plans) {
            cufft_check(hipfftSetWorkArea(pair.second, dev_workspace.get()));
        }
    }

    ~CudaTask() {
        cuda_check(hipSetDevice(device));
        for (auto &pair : plans) {
            hipfftDestroy(pair.second);
        }
    }

    void compute(const SpectrogramRequest &request) {
        cuda_check(hipSetDevice(device));
        const index_t total_input_length = request.envelope_length + (request.how_many - 1) * request.input_shift;
        IndexRange overlap = IndexRange(-request.input_offset, request.channel_length - request.input_offset).overlap(total_input_length);
        if (!overlap) {
            // all samples consist of zero-padding
            std::fill(request.maxima, request.maxima + request.how_many, ExtractedMaximum{0, 0});
            return;
        }

        info->window_length = request.window_length;
        info->spectrum_length = request.window_length / 2 + 1;
        info->envelope_length = request.envelope_length;
        info->input_shift = request.input_shift;
        info->output_bins = request.output_bins;
        info->envelope = dev_envelope.get();
        info->correctors = dev_correctors.get();
        // only supported for powers of 2
        for (info->window_length_bits = 0, info->window_length_mask = 1;
             request.window_length != info->window_length_mask; ++info->window_length_bits) {
            info->window_length_mask <<= 1;
        }
        info->window_length_mask--;

        cuda_check(hipMemcpyAsync(dev_envelope.get(), request.envelope, request.envelope_length * sizeof(real),
                                   hipMemcpyHostToDevice, stream.get()));
        cuda_check(hipMemcpyAsync(dev_correctors.get(), request.correctors,
                                   request.output_bins * sizeof(Corrector), hipMemcpyHostToDevice, stream.get()));
        cuda_check(hipMemcpyAsync(dev_info.get(), info.get(), sizeof(CudaCallbackInfo), hipMemcpyHostToDevice, stream.get()));

        SpectrogramRequest part(request);
        int how_many = 1, how_many_left = request.how_many;
        while (how_many <= how_many_left) {
            how_many *= 2;
        }
        assert(how_many > how_many_left);
        while (how_many_left > 0 && (how_many /= 2) > 0) {
            auto plan_iterator = plans.find({request.window_length, how_many});
            if (plan_iterator != plans.end()) {
                while (how_many_left >= how_many) {
                    hipfftHandle handle = plan_iterator->second;
                    part.how_many = how_many;

                    process(part, handle);

                    part.input_offset += static_cast<index_t>(request.input_shift) * static_cast<index_t>(how_many);
                    part.maxima += how_many;
                    how_many_left -= how_many;
                }
            }
        }
        cuda_check(hipStreamSynchronize(stream.get()));
    }
};

//////////////////////////////////////////////////////////////////////////////

SpectrogramCalculatorCUDA::SpectrogramCalculatorCUDA(int channel_count, const std::list<ProtoRequest> &proto_requests, int device) {
    task = std::make_shared<CudaTask>(channel_count, proto_requests, device);
}

void SpectrogramCalculatorCUDA::compute(const SpectrogramRequest &request) {
    request.assertCorrectness();
    task->compute(request);
}
