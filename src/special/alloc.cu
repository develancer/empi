/**********************************************************
 * Piotr T. Różański (c) 2015-2021                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include "PinnedArray.h"

// This file provides allocation of pinned (page-locked) host memory
// so it can be safely used in some CUDA routines.
// Each executable should link with either alloc.cpp or alloc.cu, but not both at once.

void *cuda_host_malloc(size_t length) {
    void *result;
    if (hipHostMalloc(&result, length)) {
        throw std::bad_alloc();
    }
    return result;
}

void cuda_host_free(void *pointer) {
    hipHostFree(pointer);
}
