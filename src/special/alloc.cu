/**********************************************************
 * Piotr T. Różański (c) 2015-2021                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include "PinnedArray.h"

// This file provides allocation of pinned (page-locked) host memory
// so it can be safely used in some CUDA routines.
// Each executable should link with either alloc.cpp or alloc.cu, but not both at once.

static bool CUDA_ALLOC_DISABLED = false;

void cuda_host_disable() {
    CUDA_ALLOC_DISABLED = true;
}

void *cuda_host_malloc(size_t length) {
    void *result;
    if (CUDA_ALLOC_DISABLED) {
        result = malloc(length);
        if (!result) {
            throw std::bad_alloc();
        }
    } else {
        if (hipHostMalloc(&result, length)) {
            throw std::bad_alloc();
        }
    }
    return result;
}

void cuda_host_free(void *pointer) {
    if (CUDA_ALLOC_DISABLED) {
        free(pointer);
    } else {
        hipHostFree(pointer);
    }
}
