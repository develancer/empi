#include "hip/hip_runtime.h"
/**********************************************************
 * Piotr T. Różański (c) 2015-2021                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include <algorithm>
#include <list>
#include <map>
#include <memory>
#include <set>
#include <utility>
#include <vector>
#include "Array.h"
#include "Corrector.h"
#include "CUDA.h"
#include "IndexRange.h"
#include "PinnedArray.h"
#include "WorkerCUDA.h"

char CudaException::buffer[256];

void cuda_check(hipError_t error) {
    if (error) {
        if (error == hipErrorOutOfMemory) {
            throw CudaMemoryException();
        }
        throw CudaException(error);
    }
}

void cufft_check(hipfftResult_t result) {
    if (result) {
        if (result == HIPFFT_ALLOC_FAILED) {
            throw CudaMemoryException();
        }
        throw CudaException(result);
    }
}

//////////////////////////////////////////////////////////////////////////////

class CudaCorrectorResult {
    cucomplex ft;
    cucomplex corrected;
    double norm_corrected;

public:
    __device__ CudaCorrectorResult(cucomplex ft, cucomplex corrected) : ft(ft), corrected(corrected),
                                                                        norm_corrected(corrected.x * corrected.x + corrected.y * corrected.y) {}

    // result has to be multiplied by family->value(0.0) / std::sqrt(scale);
    __device__ double amplitude() const {
        return sqrt(norm_corrected);
    }

    __device__ double energy() const {
        cucomplex corrected2 = hipCmul(corrected, corrected);
        return 0.5 * (norm_corrected + corrected2.x * ft.x + corrected2.y * ft.y);
    }

    __device__ cucomplex modulus() const {
        double factor = sqrt(energy() / norm_corrected);
        return make_hipDoubleComplex(corrected.x * factor, corrected.y * factor);
    }
};

//////////////////////////////////////////////////////////////////////////////

class CudaCorrector {
    cucomplex ft;
    double common_factor;
    double estimate_factor;

    CudaCorrector() = delete;

public:
    __device__ CudaCorrectorResult compute(cucomplex value) const {
        const bool is_value_real = std::abs(hipCimag(value)) <= 1.0e-10 * std::abs(hipCreal(value));
        if (is_value_real) {
            return CudaCorrectorResult(ft, hipCdiv(value, make_hipDoubleComplex(0.5 + hipCreal(ft) / 2, hipCimag(ft) / 2)));
        }
        const cucomplex corrected = hipCsub(value, hipCmul(hipConj(value), ft));
        return CudaCorrectorResult(ft, make_hipDoubleComplex(corrected.x * common_factor, corrected.y * common_factor));
    }
} __attribute__ ((aligned (16)));

//////////////////////////////////////////////////////////////////////////////

static __device__ __inline__ void oneReduce(volatile ExtractedMaximum &dest, const volatile ExtractedMaximum &other) {
    if (other.energy > dest.energy) {
        dest.energy = other.energy;
        dest.bin_index = other.bin_index;
    }
}

template<unsigned THREADS_IN_BLOCK>
static __device__ __inline__ void warpReduce(volatile ExtractedMaximum *sdata, unsigned tid) {
    if (THREADS_IN_BLOCK >= 64) oneReduce(sdata[tid], sdata[tid + 32]);
    if (THREADS_IN_BLOCK >= 32) oneReduce(sdata[tid], sdata[tid + 16]);
    if (THREADS_IN_BLOCK >= 16) oneReduce(sdata[tid], sdata[tid + 8]);
    if (THREADS_IN_BLOCK >= 8) oneReduce(sdata[tid], sdata[tid + 4]);
    if (THREADS_IN_BLOCK >= 4) oneReduce(sdata[tid], sdata[tid + 2]);
    if (THREADS_IN_BLOCK >= 2) oneReduce(sdata[tid], sdata[tid + 1]);
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorSingleChannelCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;
    assert(channel_count == 1);

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        real energy = correctors[i].compute(spectra[0][offset + i]).energy();
        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorConstantPhaseCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        cucomplex direction = make_hipDoubleComplex(0, 0);
        for (unsigned c = 0; c < channel_count; ++c) {
            direction = hipCfma(spectra[c][offset + i], spectra[c][offset + i], direction);
        }
        double angle = 0.5 * atan2(hipCimag(direction), hipCreal(direction));
        double sin_angle, cos_angle;
        sincos(angle, &sin_angle, &cos_angle);
        auto corrector_result = correctors[i].compute(make_hipDoubleComplex(cos_angle, sin_angle));

        double energy = 0;
        for (unsigned c = 0; c < channel_count; ++c) {
            const cucomplex v = spectra[c][offset + i];
            double cos_factor = cos(atan2(hipCimag(v), hipCreal(v)) - angle);
            energy += (hipCreal(v) * hipCreal(v) + hipCimag(v) * hipCimag(v)) * cos_factor * cos_factor;
        }
        energy *= corrector_result.energy();
        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

template<unsigned THREADS_IN_BLOCK>
static __global__ void
extractorVariablePhaseCUDA(unsigned channel_count, unsigned output_bins, cucomplex **spectra,
                           ExtractedMaximum *maxima, CudaCorrector *correctors) {
    __shared__ ExtractedMaximum sdata[THREADS_IN_BLOCK];
    const unsigned int tid = threadIdx.x;
    sdata[tid].energy = 0.0;
    sdata[tid].bin_index = 0;

    unsigned offset = blockIdx.x * output_bins;
    for (unsigned i = tid; i < output_bins; i += THREADS_IN_BLOCK) {
        // tutaj każdy wątek zbiera po wszystkich elementach którymi ma się zająć
        real energy = 0;
        for (unsigned c = 0; c < channel_count; ++c) {
            const cucomplex v = spectra[c][offset + i];
            energy += correctors[i].compute(v).energy();
        }
        if (energy > sdata[tid].energy) {
            sdata[tid].energy = energy;
            sdata[tid].bin_index = i;
        }
    }
    __syncthreads();

    // tutaj mamy już tylko tyle elementów ile wątków
    if (THREADS_IN_BLOCK >= 512) {
        if (tid < 256) oneReduce(sdata[tid], sdata[tid + 256]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 256) {
        if (tid < 128) oneReduce(sdata[tid], sdata[tid + 128]);
        __syncthreads();
    }
    if (THREADS_IN_BLOCK >= 128) {
        if (tid < 64) oneReduce(sdata[tid], sdata[tid + 64]);
        __syncthreads();
    }
    if (tid < 32) warpReduce<THREADS_IN_BLOCK>(sdata, tid);
    if (tid == 0) {
        maxima[blockIdx.x].bin_index = sdata[0].bin_index;
        maxima[blockIdx.x].energy = sdata[0].energy;
    }
}

void callExtractorKernel(const SpectrogramRequest &request, hipStream_t stream, cucomplex **spectra,
                         ExtractedMaximum *maxima, Corrector *correctors) {
    const unsigned threads_in_block = 64;
    if (request.extractor == extractorVariablePhase) {
        extractorVariablePhaseCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
        (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else if (request.extractor == extractorConstantPhase) {
        extractorConstantPhaseCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
        (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else if (request.extractor == extractorSingleChannel) {
        extractorSingleChannelCUDA<threads_in_block><<<request.how_many, threads_in_block, 0, stream>>>
                (request.channel_count, request.output_bins, spectra, maxima, reinterpret_cast<CudaCorrector *>(correctors));
    } else {
        fprintf(stderr, "ERROR: unsupported extractor\n");
        exit(1);
    }
}

void *cuda_dev_malloc(size_t length) {
    void *result;
    cuda_check(hipMalloc(&result, length));
    return result;
}

void cuda_dev_free(void *pointer) {
    cuda_check(hipFree(pointer));
}

template<typename T>
T *cuda_dev_alloc(size_t length) {
    return reinterpret_cast<T *>(cuda_dev_malloc(sizeof(T) * length));
}

template<typename T>
Array1D<T> cuda_dev_array_1d(size_t n) {
    return Array1D<T>(n, cuda_dev_alloc<T>, cuda_dev_free);
}

template<typename T>
Array2D<T> cuda_dev_array_2d(size_t n, size_t m) {
    return Array2D<T>(n, m, cuda_dev_alloc<T>, cuda_dev_free);
}

std::shared_ptr<ihipStream_t> cuda_create_stream() {
    hipStream_t stream;
    cuda_check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    return std::shared_ptr<ihipStream_t>(stream, hipStreamDestroy);
}

//////////////////////////////////////////////////////////////////////////////

class CudaTask {
    std::map<std::pair<int, int>, hipfftHandle> plans;

    std::shared_ptr<CudaCallbackInfo> info;
    std::shared_ptr<CudaCallbackInfo> dev_info;
    Array1D<real> dev_envelope;
    Array1D<Corrector> dev_correctors;

    std::shared_ptr<ihipStream_t> stream;

    Array1D<char> dev_workspace;
    Array1D<real> dev_input;
    Array2D<cucomplex> outputs;
    Array1D<cucomplex *> dev_outputs;
    Array1D<ExtractedMaximum> dev_maxima;

    void process(const SpectrogramRequest &request, hipfftHandle handle) {
        index_t total_input_length = request.envelope_length + (request.how_many - 1) * request.input_shift;
        IndexRange overlap = IndexRange(-request.input_offset, request.channel_length - request.input_offset).overlap(total_input_length);
        if (!overlap) {
            // all samples consist of zero-padding
            std::fill(request.maxima, request.maxima + request.how_many, ExtractedMaximum{0, 0});
            return;
        }

        for (int c = 0; c < request.channel_count; ++c) {
            if (overlap.first_index) {
                cuda_check(hipMemsetAsync(dev_input.get(), 0, overlap.first_index * sizeof(real), stream.get()));
            }
            cuda_check(hipMemcpyAsync(dev_input.get() + overlap.first_index,
                                       request.data[c] + request.input_offset + overlap.first_index,
                                       (overlap.end_index - overlap.first_index) * sizeof(real), hipMemcpyHostToDevice,
                                       stream.get()));
            if (overlap.end_index < total_input_length) {
                cuda_check(hipMemsetAsync(dev_input.get() + overlap.end_index, 0,
                                           (total_input_length - overlap.end_index) * sizeof(real), stream.get()));
            }
            cufft_check(hipfftExecD2Z(handle, dev_input.get(), outputs[c]));
        }
        callExtractorKernel(request, stream.get(), dev_outputs.get(), dev_maxima.get(), dev_correctors.get());
        cuda_check(hipMemcpyAsync(request.maxima, dev_maxima.get(), request.how_many * sizeof(ExtractedMaximum),
                                   hipMemcpyDeviceToHost, stream.get()));
    }

public:
    CudaTask(int channel_count, const std::list<ProtoRequest> &proto_requests) {
        int max_envelope_length = 0;
        int max_output_bins = 0;
        int max_window_length = 0;
        for (const auto &proto_request : proto_requests) {
            max_envelope_length = std::max(max_envelope_length, proto_request.envelope_length);
            max_output_bins = std::max(max_output_bins, proto_request.output_bins);
            max_window_length = std::max(max_window_length, proto_request.window_length);
        }

        stream = cuda_create_stream();

        info.reset(cuda_host_alloc<CudaCallbackInfo>(1), cuda_host_free);
        dev_info.reset(cuda_dev_alloc<CudaCallbackInfo>(1), cuda_dev_free);
        dev_envelope = cuda_dev_array_1d<real>(max_envelope_length);
        dev_correctors = cuda_dev_array_1d<Corrector>(max_output_bins);
        dev_outputs = cuda_dev_array_1d<cucomplex *>(channel_count);

        size_t free_memory, total_memory;
        cuda_check(hipMemGetInfo(&free_memory, &total_memory));

        size_t max_dev_input_length = 0;
        size_t max_dev_output_length = 0;
        size_t max_work_size = 0;
        int max_how_many = 0;
        std::map<int, std::set<int>> how_many_for_various_windows;
        for (const auto &proto_request : proto_requests) {
            int window_length = proto_request.window_length;
            int spectrum_length = window_length / 2 + 1;

            for (int how_many = 1; how_many <= proto_request.how_many; how_many *= 2) {
                size_t dev_input_length = static_cast<size_t>(proto_request.envelope_length)
                                          + static_cast<size_t>(how_many - 1) * static_cast<size_t>(proto_request.input_shift);
                size_t dev_output_length = static_cast<size_t>(how_many) * static_cast<size_t>(proto_request.output_bins);
                size_t work_size;
                cufft_check(hipfftEstimateMany(
                        1, &window_length,
                        &window_length, 1, window_length,
                        &spectrum_length, 1, spectrum_length,
                        HIPFFT_D2Z, how_many, &work_size
                ));

                size_t memory_needed = work_size
                                       + sizeof(double) * dev_input_length
                                       + sizeof(cucomplex) * static_cast<size_t>(channel_count) * dev_output_length
                                       + sizeof(ExtractedMaximum) * static_cast<size_t>(how_many);
                if (memory_needed > free_memory) {
                    if (how_many == 1) {
                        throw std::runtime_error("not enough CUDA memory");
                    }
                    break;
                }

                max_dev_input_length = std::max(max_dev_input_length, dev_input_length);
                max_dev_output_length = std::max(max_dev_output_length, dev_output_length);
                max_work_size = std::max(max_work_size, work_size);
                max_how_many = std::max(max_how_many, how_many);

                how_many_for_various_windows[window_length].insert(how_many);
            }
        }

        dev_input = cuda_dev_array_1d<double>(max_dev_input_length);

        outputs = cuda_dev_array_2d<cucomplex>(channel_count, max_dev_output_length);
        cuda_check(hipMemcpy(dev_outputs.get(), outputs.get(), channel_count * sizeof(cucomplex *), hipMemcpyHostToDevice));

        dev_maxima = cuda_dev_array_1d<ExtractedMaximum>(max_how_many);

        max_work_size = 0; // this will now be calculated more precisely
        for (const auto &pair : how_many_for_various_windows) {
            int window_length = pair.first;
            int spectrum_length = window_length / 2 + 1;
            for (int how_many : pair.second) {
                hipfftHandle handle;
                size_t work_size = 0;
                cufft_check(hipfftCreate(&handle));
                cufft_check(hipfftSetAutoAllocation(handle, 0));
                cufft_check(hipfftMakePlanMany(
                        handle, 1, &window_length,
                        &window_length, 1, window_length,
                        &spectrum_length, 1, spectrum_length,
                        HIPFFT_D2Z, how_many, &work_size
                ));
                max_work_size = std::max(max_work_size, work_size);
                associateCallbackWithPlan(handle, dev_info.get());
                cufft_check(hipfftSetStream(handle, stream.get()));
                plans.emplace(std::make_pair(window_length, how_many), std::move(handle));
            }
        }

        dev_workspace = cuda_dev_array_1d<char>(max_work_size);
        for (auto &pair : plans) {
            cufft_check(hipfftSetWorkArea(pair.second, dev_workspace.get()));
        }
    }

    ~CudaTask() {
        for (auto &pair : plans) {
            hipfftDestroy(pair.second);
        }
    }

    void compute(const SpectrogramRequest &request) {
        const index_t total_input_length = request.envelope_length + (request.how_many - 1) * request.input_shift;
        IndexRange overlap = IndexRange(-request.input_offset, request.channel_length - request.input_offset).overlap(total_input_length);
        if (!overlap) {
            // all samples consist of zero-padding
            std::fill(request.maxima, request.maxima + request.how_many, ExtractedMaximum{0, 0});
            return;
        }

        info->window_length = request.window_length;
        info->spectrum_length = request.window_length / 2 + 1;
        info->envelope_length = request.envelope_length;
        info->input_shift = request.input_shift;
        info->output_bins = request.output_bins;
        info->envelope = dev_envelope.get();
        info->correctors = dev_correctors.get();
        // only supported for powers of 2
        for (info->window_length_bits = 0, info->window_length_mask = 1;
             request.window_length != info->window_length_mask; ++info->window_length_bits) {
            info->window_length_mask <<= 1;
        }
        info->window_length_mask--;

        cuda_check(hipMemcpyAsync(dev_envelope.get(), request.envelope, request.envelope_length * sizeof(real),
                                   hipMemcpyHostToDevice, stream.get()));
        cuda_check(hipMemcpyAsync(dev_correctors.get(), request.correctors,
                                   request.output_bins * sizeof(Corrector), hipMemcpyHostToDevice, stream.get()));
        cuda_check(hipMemcpyAsync(dev_info.get(), info.get(), sizeof(CudaCallbackInfo), hipMemcpyHostToDevice, stream.get()));

        SpectrogramRequest part(request);
        int how_many = 1, how_many_left = request.how_many;
        while (how_many <= how_many_left) {
            how_many *= 2;
        }
        assert(how_many > how_many_left);
        while (how_many_left > 0 && (how_many /= 2) > 0) {
            auto plan_iterator = plans.find({request.window_length, how_many});
            if (plan_iterator != plans.end()) {
                while (how_many_left >= how_many) {
                    hipfftHandle handle = plan_iterator->second;
                    part.how_many = how_many;

                    process(part, handle);

                    part.input_offset += static_cast<index_t>(request.input_shift) * static_cast<index_t>(how_many);
                    part.maxima += how_many;
                    how_many_left -= how_many;
                }
            }
        }
        cuda_check(hipStreamSynchronize(stream.get()));
    }
};

//////////////////////////////////////////////////////////////////////////////

WorkerCUDA::WorkerCUDA(int channel_count, const std::list<ProtoRequest> &proto_requests) {
    task = std::make_shared<CudaTask>(channel_count, proto_requests);
}

void WorkerCUDA::compute(const SpectrogramRequest &request) {
    request.assertCorrectness();
    task->compute(request);
}
