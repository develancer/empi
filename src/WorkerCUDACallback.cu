/**********************************************************
 * Piotr T. Różański (c) 2015-2021                        *
 *   Enhanced Matching Pursuit Implementation (empi)      *
 * See README.md and LICENCE for details.                 *
 **********************************************************/
#include "CUDA.h"

static __device__  real myInputCallback(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
    CudaCallbackInfo *info = reinterpret_cast<CudaCallbackInfo *>(callerInfo);
    size_t index_in_batch = offset & info->window_length_mask;
    real ret = (index_in_batch < info->envelope_length)
               ? ((real *) dataIn)[(offset >> info->window_length_bits) * info->input_shift + index_in_batch] * info->envelope[index_in_batch]
               : 0;
    return ret;
}

static __device__  void
myOutputCallback(void *dataOut, size_t offset, cucomplex element, void *callerInfo, void *sharedPtr) {
    CudaCallbackInfo *info = reinterpret_cast<CudaCallbackInfo *>(callerInfo);
    size_t index_in_batch = offset % info->spectrum_length;
    if (index_in_batch < info->output_bins) {
        ((cucomplex *) dataOut)[offset / info->spectrum_length * info->output_bins + index_in_batch] = element;
    }
}

static __device__ hipfftCallbackLoadD myInputPtr = myInputCallback;

static __device__ hipfftCallbackStoreZ myOutputPtr = myOutputCallback;

static hipfftCallbackLoadD hostCopyOfInputCallback = nullptr;

static hipfftCallbackStoreZ hostCopyOfOutputCallback = nullptr;

void associateCallbackWithPlan(hipfftHandle plan, CudaCallbackInfo *dev_info) {
    if (!hostCopyOfInputCallback) {
        cuda_check(hipMemcpyFromSymbol(&hostCopyOfInputCallback, HIP_SYMBOL(myInputPtr), sizeof(hostCopyOfInputCallback)));
    }
    if (!hostCopyOfOutputCallback) {
        cuda_check(hipMemcpyFromSymbol(&hostCopyOfOutputCallback, HIP_SYMBOL(myOutputPtr), sizeof(hostCopyOfOutputCallback)));
    }
    cufft_check(hipfftXtSetCallback(plan, (void **) &hostCopyOfInputCallback, HIPFFT_CB_LD_REAL_DOUBLE, (void **) &dev_info));
    cufft_check(hipfftXtSetCallback(plan, (void **) &hostCopyOfOutputCallback, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void **) &dev_info));
}
